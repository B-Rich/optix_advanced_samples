#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable( float3,      cutoff_color, , );
rtDeclareVariable( float,       frequency, , );

rtDeclareVariable(float3,  shading_normal, attribute shading_normal, ); 
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,  front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, PerRayData_radiance prd_in )
{
    optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
    PerRayData_radiance prd;
    prd.depth = prd_in.depth+1;
    prd.seed = prd_in.seed;

    rtTrace( top_object, ray, prd );
    return prd.result;
}

RT_PROGRAM void closest_hit_radiance()
{

    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    const float z1 = rnd( prd_radiance.seed );
    const float z2 = rnd( prd_radiance.seed );
    
    float3 traced_color = cutoff_color;
    if (prd_radiance.depth < max_depth) {
        float3 w_in;
        cosine_sample_hemisphere( z1, z2, w_in );
        optix::Onb onb( ffnormal );
        onb.inverse_transform( w_in );
        const float3 fhp = rtTransformPoint( RT_OBJECT_TO_WORLD, front_hit_point );
        traced_color = TraceRay( fhp, w_in, prd_radiance );
    }
    
    const float u = frequency * texcoord.x;
    const float uu = u - floorf( u );
    const float v = frequency * texcoord.y;
    const float vv = v - floorf( v );
    const float linewidth = 0.04f;
    const float halflinewidth = 0.5f*linewidth;
    const float gridval = 1.0f - fmaxf( 
        smoothstep( 0.5f - linewidth, 0.5f - halflinewidth, uu ) - smoothstep( 0.5f + halflinewidth, 0.5f + linewidth, uu ),
        smoothstep( 0.5f - linewidth, 0.5f - halflinewidth, vv ) - smoothstep( 0.5f + halflinewidth, 0.5f + linewidth, vv )
        );
    const float3 Kd = make_float3( gridval );
    prd_radiance.result = Kd * traced_color;

}

