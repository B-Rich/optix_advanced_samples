#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "prd.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, PerRayData_radiance prd_in )
{
  optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
  PerRayData_radiance prd;
  prd.depth = prd_in.depth+1;
  prd.seed = prd_in.seed;

  rtTrace( top_object, ray, prd );
  return prd.result;
}

static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

static __device__ __inline__ float fresnel( float cos_theta_i, float cos_theta_t, float eta )
{
    const float rs = ( cos_theta_i - cos_theta_t*eta ) / 
                     ( cos_theta_i + eta*cos_theta_t );
    const float rp = ( cos_theta_i*eta - cos_theta_t ) /
                     ( cos_theta_i*eta + cos_theta_t );

    return 0.5f * ( rs*rs + rp*rp );
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
    const float3 w_out = -ray.direction;
    float3 normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float cos_theta_i = optix::dot( w_out, normal );

    float eta;
    float3 attenuation = make_float3( 1.0f );
    if( cos_theta_i > 0.0f ) {
        // Ray is entering 
        eta = refraction_index;  // Note: does not handle nested dielectrics
    } else {
        // Ray is exiting.
        attenuation = exp(extinction_constant * t_hit);
        eta         = 1.0f / refraction_index;
        cos_theta_i = -cos_theta_i;
        normal      = -normal;
    }

    float3 w_t;
    const bool tir           = !optix::refract( w_t, -w_out, normal, eta );

    const float cos_theta_t  = -optix::dot( normal, w_t );
    const float R            = tir  ?
                               1.0f :
                               fresnel( cos_theta_i, cos_theta_t, eta );

    float3 traced_color = cutoff_color;

    const float z = rnd( prd_radiance.seed );
    if( z <= R ) {
        // Reflect
        if (prd_radiance.depth < max_depth) {
            const float3 w_in = optix::reflect( -w_out, normal ); 
            const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
            traced_color = TraceRay( fhp, w_in, prd_radiance );
        }
        prd_radiance.result = reflection_color*attenuation*traced_color;
    } else {
        // Refract
        if (prd_radiance.depth < max_depth) {
            const float3 w_in = w_t;
            const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
            traced_color = TraceRay( bhp, w_in, prd_radiance );
        }
        prd_radiance.result = refraction_color*attenuation*traced_color;
    }

}


