#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

struct PerRayData_radiance
{
  float3 result;
  int depth;
  unsigned int seed;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, PerRayData_radiance prd_in )
{
  optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
  PerRayData_radiance prd;
  prd.depth = prd_in.depth+1;
  prd.seed = prd_in.seed;

  rtTrace( top_object, ray, prd );
  return prd.result;
}

static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
  // intersection vectors
  const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
  const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);

  // Refract and check for total internal reflection
  float3 transmission_direction;
  const bool tir = !( refract(transmission_direction, ray.direction, n, refraction_index) );

  // check for external or internal reflection
  const float cos_theta_i = dot(ray.direction, n);
  float cos_theta = 0.0f;
  if ( !tir ) {
      if (cos_theta_i < 0.0f) {
          cos_theta = -cos_theta_i;
      } else {
          cos_theta = dot(transmission_direction, n);
      }
  }

  const float reflection_weight = tir ? 1.0f : fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);
  const float P = /*reflection_weight*/ tir ? 1.0f : 0.5f;
  const bool do_reflection = ( rnd( prd_radiance.seed ) <  P );

  float3 result = make_float3(0.0f);
  float3 color = cutoff_color;

  if ( do_reflection )  {
      if (prd_radiance.depth < max_depth) {
          const float3 r = reflect(ray.direction, n);
          color = TraceRay( fhp, r, prd_radiance );
      }
      result += ( reflection_weight / P ) * reflection_color * color;
  }
  else {  // refraction
      if (prd_radiance.depth < max_depth) {
          color = TraceRay(bhp, transmission_direction, prd_radiance);
      }
      result += ( ( 1.0f - reflection_weight ) / ( 1 - P ) )* refraction_color * color;
  }

  if( cos_theta_i > 0 ) {
    // Beer's Law attenuation when exiting surface
    result *= exp(extinction_constant * t_hit);
  }

  prd_radiance.result = result;
}


