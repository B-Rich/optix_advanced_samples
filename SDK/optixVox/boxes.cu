#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

#include "intersection_refinement.h"

using namespace optix;

// Compressed 8-bit indices as in VOX format.  We expand these into floating point coords during intersection.
rtBuffer< optix::uchar4 > box_buffer;

rtBuffer< optix::uchar4 > palette_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable( float3, back_hit_point, attribute back_hit_point, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(uchar4, geometry_color, attribute geometry_color, ); 

static __device__ float3 boxnormal(float3 boxmin, float3 boxmax, float t)
{
    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 neg = make_float3(t==t0.x?1:0, t==t0.y?1:0, t==t0.z?1:0);
    float3 pos = make_float3(t==t1.x?1:0, t==t1.y?1:0, t==t1.z?1:0);
    return pos-neg;
}

RT_PROGRAM void intersect( int primId )
{
    // Expand cell in unit box
    const uchar4 b = box_buffer[primId];
    const float3 inv_box_dims = make_float3( 1.0f ) / make_float3( 255.0f );
    const float3 boxmin = make_float3( b.x, b.y, b.z ) * inv_box_dims;
    const float3 boxmax = boxmin + inv_box_dims;

    float3 t0 = (boxmin - ray.origin)/ray.direction;
    float3 t1 = (boxmax - ray.origin)/ray.direction;
    float3 near = fminf(t0, t1);
    float3 far = fmaxf(t0, t1);
    float tmin = fmaxf( near );
    float tmax = fminf( far );

    if(tmin <= tmax) {
        bool check_second = true;
        if( rtPotentialIntersection( tmin ) ) {
            int color_index = (int)box_buffer[primId].w;
            geometry_color = palette_buffer[ color_index ];
            shading_normal = geometric_normal = boxnormal( boxmin, boxmax, tmin );

            // TODO: refine
            const float3 p = ray.origin + tmin*ray.direction;
            back_hit_point = front_hit_point = p;

            if(rtReportIntersection(0))
                check_second = false;
        } 
        if(check_second) {
            if( rtPotentialIntersection( tmax ) ) {
                int color_index = (int)box_buffer[primId].w;
                geometry_color = palette_buffer[ color_index ];
                shading_normal = geometric_normal = boxnormal( boxmin, boxmax, tmax );

                // TODO: refine
                const float3 p = ray.origin + tmax*ray.direction;
                back_hit_point = front_hit_point = p;

                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void bounds (int primId, float result[6])
{
    const uchar4 b = box_buffer[primId];
    const float3 inv_box_dims = make_float3( 1.0f ) / make_float3( 255.0f );
    const float3 boxmin = make_float3( b.x, b.y, b.z ) * inv_box_dims;
    const float3 boxmax = boxmin + inv_box_dims;

    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set( boxmin, boxmax );
}

